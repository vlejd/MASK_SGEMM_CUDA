#include <hip/hip_fp16.h>
#include <iostream>
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <vector>
#include <hipblas.h>
#include <hip/hip_runtime.h>


#define cudaCheck(err) (cudaCheckInternal(err, __FILE__, __LINE__))

void cudaCheckInternal(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
               hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};

const std::string errLogFile = "matrixValidationFailure.txt";
const std::string dbgLogFile = "matrixValidationDebug.txt";

class Problem_InstanceFP16
{
public:
    int M, N, K;
    int seed;
    __half *hA, *hB, *hBt, *hC, *hC_ref;
    __half *dA, *dB, *dBt, *dC, *dC_ref;
    int *hMask;
    int *dMask;
    float density;
    void get_result();
    void get_result_ref();
    Problem_InstanceFP16(int M, int N, int K, float density, int seed = 0);
    ~Problem_InstanceFP16();
};

void Problem_InstanceFP16::get_result()
{
    cudaCheck(hipMemcpy(this->hC, this->dC, sizeof(__half) * this->M * this->N, hipMemcpyDeviceToHost));
}

void Problem_InstanceFP16::get_result_ref()
{
    cudaCheck(hipMemcpy(this->hC_ref, this->dC_ref, sizeof(__half) * this->M * this->N, hipMemcpyDeviceToHost));
}

Problem_InstanceFP16::~Problem_InstanceFP16()
{
    free(this->hA);
    free(this->hB);
    free(this->hBt);
    free(this->hC);
    hipFree(this->dA);
    hipFree(this->dB);
    hipFree(this->dBt);
    hipFree(this->dC);
}

void zero_init_matrix(__half *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = 0.0;
    }
}

void randomize_matrix(__half *mat, int N, int seed)
{
    // NOTICE: Use gettimeofday instead of srand((unsigned)time(NULL)); the time
    // precision is too low and the same random number is generated.
    srand(seed);
    for (int i = 0; i < N; i++)
    {
        //float tmp = (float)(rand() % 5) + 0.1 * (rand() % 5);
        //tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
        float tmp = (float)(rand() % 4);
        mat[i] = __float2half(tmp);
    }
}

void generate_mask(int *mask, int M, int N, float density, int seed)
{
    srand(seed);
    for (int i = 0; i < M * N; i++)
    {
        mask[i] = (rand() % 100 < density * 100) ? 1 : 0;
    }
}

Problem_InstanceFP16::Problem_InstanceFP16(int M, int N, int K, float density, int seed)
{
    this->M = M;
    this->N = N;
    this->K = K;
    this->density = density;
    this->seed = seed;

    this->hA = (__half *)malloc(sizeof(__half) * this->M * this->K);
    this->hB = (__half *)malloc(sizeof(__half) * this->K * this->N);
    this->hBt = (__half *)malloc(sizeof(__half) * this->K * this->N);
    this->hC = (__half *)malloc(sizeof(__half) * this->M * this->N);
    this->hC_ref = (__half *)malloc(sizeof(__half) * this->M * this->N);
    this->hMask = (int *)malloc(sizeof(int) * this->K * this->N);

    randomize_matrix(this->hA, this->M * this->K, this->seed);
    randomize_matrix(this->hB, this->K * this->N, this->seed+1);
    zero_init_matrix(this->hC, this->M * this->N);
    zero_init_matrix(this->hC_ref, this->M * this->N);
    generate_mask(this->hMask, this->K, this->N, this->density, this->seed+2);
    // apply_mask(this->hB, this->hMask, this->K, this->N);
    // transpose(this->hB, this->hBt, this->K, this->N);

    cudaCheck(hipMalloc((void **)&this->dA, sizeof(__half) * this->M * this->K));
    cudaCheck(hipMalloc((void **)&this->dB, sizeof(__half) * this->K * this->N));
    cudaCheck(hipMalloc((void **)&this->dBt, sizeof(__half) * this->K * this->N));
    cudaCheck(hipMalloc((void **)&this->dC, sizeof(__half) * this->M * this->N));
    cudaCheck(hipMalloc((void **)&this->dC_ref, sizeof(__half) * this->M * this->N));

    cudaCheck(hipMemcpy(this->dA, this->hA, sizeof(__half) * this->M * this->K, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(this->dB, this->hB, sizeof(__half) * this->K * this->N, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(this->dBt, this->hBt, sizeof(__half) * this->K * this->N, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(this->dC, this->hC, sizeof(__half) * this->M * this->N, hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(this->dC_ref, this->hC_ref, sizeof(__half) * this->M * this->N, hipMemcpyHostToDevice));
}

template <typename T>
void print_matrix(const T *A, int M, int N, std::ofstream &fs)
{
  int i;
  fs << std::setprecision(2)
     << std::fixed; // Set floating-point precision and fixed notation
  fs << "[";
  for (i = 0; i < M && i < 32; i++)
  {
    for (int j = 0; j < N && j < 32; j++)
    {
      fs << __half2float(A[i * N + j]);
      if (j != N - 1)
        fs << ", ";
    }
    if (i != M - 1)
      fs << ";\n";
  }
  fs << "]\n";
}

void log_matrix_data(const std::string &fileName, const Problem_InstanceFP16 &pi)
{
    std::ofstream fs;
    fs.open(fileName);
    fs << "A:\n";
    print_matrix(pi.hA, pi.M, pi.K, fs);
    fs << "B:\n";
    print_matrix(pi.hB, pi.K, pi.N, fs);
    fs << "Bt:\n";
    print_matrix(pi.hBt, pi.K, pi.N, fs);
    fs << "Mask:\n";
    print_matrix(pi.hMask, pi.K, pi.N, fs);
    fs << "C:\n";
    print_matrix(pi.hC, pi.M, pi.N, fs);
    fs << "Should:\n";
    print_matrix(pi.hC_ref, pi.M, pi.N, fs);
};

// runCublasFP16
void runCublasFP16(hipblasHandle_t handle, Problem_InstanceFP16 &pi, bool ref)
{
    float alpha = 1.0f;
    float beta = 0.0f;
    // cublasGemmEx
    __half * result_pointer;
    if (ref){
        result_pointer = pi.dC_ref;
    } else {
        result_pointer = pi.dC;
    }
    hipblasStatus_t status = hipblasGemmEx(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        pi.N, pi.M, pi.K,
        &alpha,
        pi.dB, HIP_R_16F, pi.N,
        pi.dA, HIP_R_16F, pi.K,
        &beta,
        result_pointer, HIP_R_16F, pi.N,
        HIPBLAS_COMPUTE_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        std::cerr << "CUBLAS error: " << status << std::endl;
    }
}

void run_kernel_fp16(int kernel_num, Problem_InstanceFP16 &pi, hipblasHandle_t handle, bool ref)
{
    switch (kernel_num)
    {
    case 0:
        runCublasFP16(handle, pi, ref);
        break;
    default:
        throw std::invalid_argument("Unknown kernel number");
    }
}

bool verify_result(Problem_InstanceFP16 &pi)
{
    double diff = 0.0;
    int i;
    for (i = 0; i < pi.N * pi.M; i++)
    {
        float a = __half2float(pi.hC[i]);
        float b = __half2float(pi.hC_ref[i]);
        diff = std::fabs(a - b);

        if (diff > 0.01)
        {
            std::cout << "Divergence! Should " << a << ", Is " << b
                      << " (Diff " << diff << ") at " << i << std::endl;
            return false;
        }
    }
    return true;
}

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        std::cerr << "Please select a kernel" << std::endl;
        exit(EXIT_FAILURE);
    }

    int kernel_num = std::stoi(argv[1]);

    int deviceIdx = 0;
    if (getenv("DEVICE") != NULL)
    {
        deviceIdx = atoi(getenv("DEVICE"));
    }
    cudaCheck(hipSetDevice(deviceIdx));

    printf("Running kernel %d on device %d.\n", kernel_num, deviceIdx);

    hipblasHandle_t handle;
    if (hipblasCreate(&handle))
    {
        std::cerr << "Create cublas handle error." << std::endl;
        exit(EXIT_FAILURE);
    };

    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    // cuBLAS FLOPs ceiling is reached at 8192
    // std::vector<int> SIZE = {1<<10, 1<<11, 1<<12, 1<<13};
    // std::vector<int> SIZE = {1 << 12};
    std::vector<int> SIZE = {1<<12};
    int M = 1;

    // GEMM input parameters, C=α*AB+β*C

    float density = 0.25;
    int repeat_times = 50;
    for (int size : SIZE)
    {
        // TODO generate multiple problems and cycle through them.
        Problem_InstanceFP16 pi(M, size, size, density, 42);

        std::cout << "dimensions(m,n,k) " << pi.M << "," << pi.K << "," << pi.N << std::endl;
        run_kernel_fp16(0, pi, handle, true);
        run_kernel_fp16(kernel_num, pi, handle, false);

        
        cudaCheck(hipDeviceSynchronize());
        cudaCheck(hipGetLastError()); // Check for async errors during kernel run
        pi.get_result();
        pi.get_result_ref();
        
        if (!verify_result(pi))
        {
            std::cout << "=============================" << std::endl;
            std::cout << "Different result than cuBBLAS" << std::endl;
            std::cout << "=============================" << std::endl;
            std::cout << " Logging faulty output into " << errLogFile << "\n";
            log_matrix_data(errLogFile, pi);
        }
        else
        {
            std::cout << " Logging debug output into " << dbgLogFile << "\n";
            log_matrix_data(dbgLogFile, pi);
        }
        
        hipEventRecord(beg);
        for (int j = 0; j < repeat_times; j++)
        {
            // TODO cycle the problem instances
            run_kernel_fp16(kernel_num, pi, handle, false);
        }

        hipEventRecord(end);
        hipEventSynchronize(beg);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, beg, end);
        elapsed_time /= 1000.; // Convert to seconds

        long flops = 2 * pi.M * pi.N * pi.K;
        printf(
            "Average elapsed time: (%7.6f) s, performance: (%7.1f) GFLOPS. size: "
            "(%ldX%ldX%ld).\n\n",
            elapsed_time / repeat_times,
            (repeat_times * flops * 1e-9) / elapsed_time, pi.M, pi.K, pi.N);
        fflush(stdout);
    }
    hipblasDestroy(handle);

    return 0;
};